#include "hip/hip_runtime.h"
/*
 ==============================================================================

 RPGPUForceAtlas2.cu
 Copyright © 2016, 2017, 2018  G. Brinkmann

 This file is part of graph_viewer.

 graph_viewer is free software: you can redistribute it and/or modify
 it under the terms of version 3 of the GNU Affero General Public License as
 published by the Free Software Foundation.

 graph_viewer is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU Affero General Public License for more details.

 You should have received a copy of the GNU Affero General Public License
 along with graph_viewer.  If not, see <https://www.gnu.org/licenses/>.

 ==============================================================================
*/

// Reading: May 21st

#include <stdio.h>
#include <fstream>
#include <chrono>
#include <algorithm>
#include "time.h"

#include "RPGPUForceAtlas2.hpp"
#include "RPBHFA2LaunchParameters.cuh"
#include "RPBHKernels.cuh"
#include "RPFA2Kernels.cuh"
#include "../common/RPTypeDefs.hpp"

namespace RPGraph
{
    CUDAForceAtlas2::CUDAForceAtlas2(GraphLayout &layout, bool use_barneshut,
                                     bool strong_gravity, float gravity,
                                     float scale, bool randomize, bool use_linlog)
    : ForceAtlas2(layout, use_barneshut, strong_gravity, gravity, scale, randomize, use_linlog)
    {
        /**
         * Device count refers to how many discrete GPUs are available.
         */
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        if (deviceCount == 0)
        {
            fprintf(stderr, "error: No CUDA devices found.\n");
            exit(EXIT_FAILURE);
        }

        // Host initialization and setup //
        nbodies = layout.graph.num_nodes();
        nedges  = layout.graph.num_edges();

        /* float2 is a 2 dimensional vector alias. */
        body_pos = (float2 *)malloc(sizeof(float2) * layout.graph.num_nodes());
        body_mass = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        /* source and target for every edge */
        sources  = (int *)  malloc(sizeof(int)   * layout.graph.num_edges());
        targets  = (int *)  malloc(sizeof(int)   * layout.graph.num_edges());
        /* force at x and y??? for each node, and previous force to calculate 
         * the change in force (what did we call this again?) */
        fx       = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        fy       = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        fx_prev  = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        fy_prev  = (float *)malloc(sizeof(float) * layout.graph.num_nodes());

		// TODO: Why not just use memset?
        for (contiguous_nid_t n = 0; n < layout.graph.num_nodes(); ++n)
        {
            body_pos[n] = {layout.getX(n), layout.getY(n)}; /// What type of initialization is this?
            body_mass[n] = ForceAtlas2::mass(n);
            fx[n] = 0.0;
            fy[n] = 0.0;
            fx_prev[n] = 0.0;
            fy_prev[n] = 0.0;
        }

        int cur_sources_idx = 0;
        int cur_targets_idx = 0;

        // Initialize the sources and targets arrays with edge-data.
        for (contiguous_nid_t source_id = 0; source_id < layout.graph.num_nodes(); ++source_id)
        {
            for (contiguous_nid_t target_id : layout.graph.neighbors_with_geq_id(source_id))
            {
                sources[cur_sources_idx++] = source_id;
                targets[cur_targets_idx++] = target_id;
            }
        }

        // GPU initialization and setup //
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        if (deviceProp.warpSize != WARPSIZE)
        {
            printf("Warpsize of device is %d, but we anticipated %d\n", deviceProp.warpSize, WARPSIZE);
            exit(EXIT_FAILURE);

        }
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoundingBoxKernel), hipFuncCachePreferShared);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(TreeBuildingKernel), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel1), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel2), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel), hipFuncCachePreferShared);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernel), hipFuncCachePreferL1);
#if __CUDA_ARCH__ < 300
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferL1);
#endif
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(DisplacementKernel), hipFuncCachePreferL1);

        hipGetLastError();  // reset error value

        // Allocate space on device.
        mp_count = deviceProp.multiProcessorCount;
        max_threads_per_block = deviceProp.maxThreadsPerBlock;

        /* Why is nnodes set to 2*nbodies? */
        nnodes = std::max(2 * nbodies, mp_count * max_threads_per_block);

        // Round up to next multiple of WARPSIZE
        while ((nnodes & (WARPSIZE-1)) != 0) nnodes++;
        nnodes--;

        // child stores structure of the quadtree. values point to IDs.
        cudaCatchError(hipMalloc((void **)&childl,  sizeof(int)   * (nnodes+1) * 4));

        // the following properties, for each node in the quadtree (both internal and leaf)
        cudaCatchError(hipMalloc((void **)&body_massl,   sizeof(float) * nbodies));
        cudaCatchError(hipMalloc((void **)&node_massl,   sizeof(float) * (nnodes+1)));
        cudaCatchError(hipMalloc((void **)&body_posl,sizeof(float2) * nbodies));
        cudaCatchError(hipMalloc((void **)&node_posl,    sizeof(float2) * (nnodes+1)));
        // count contains the number of nested nodes for each node in quadtree
        cudaCatchError(hipMalloc((void **)&countl,  sizeof(int)   * (nnodes+1)));
        // start contains ...
        cudaCatchError(hipMalloc((void **)&startl,  sizeof(int)   * (nnodes+1)));
        cudaCatchError(hipMalloc((void **)&sortl,   sizeof(int)   * (nnodes+1)));


        cudaCatchError(hipMalloc((void **)&sourcesl,sizeof(int)   * (nedges)));
        cudaCatchError(hipMalloc((void **)&targetsl,sizeof(int)   * (nedges)));
        cudaCatchError(hipMalloc((void **)&fxl,     sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fyl,     sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fx_prevl,sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fy_prevl,sizeof(float) * (nbodies)));

        // Used for reduction in BoundingBoxKernel
        cudaCatchError(hipMalloc((void **)&maxxl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&maxyl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&minxl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&minyl,   sizeof(float) * mp_count * FACTOR1));

        // Used for reduction in SpeedKernel
        cudaCatchError(hipMalloc((void **)&swgl,    sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&etral,   sizeof(float) * mp_count * FACTOR1));

        // Copy host data to device.
        cudaCatchError(hipMemcpy(body_massl, body_mass, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(body_posl,  body_pos,  sizeof(float2) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(sourcesl, sources, sizeof(int) * nedges, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(targetsl, targets, sizeof(int) * nedges, hipMemcpyHostToDevice));

        // cpy fx, fy , fx_prevl, fy_prevl so they are all initialized to 0 in device memory.
        cudaCatchError(hipMemcpy(fxl, fx,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fyl, fy,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fx_prevl, fx_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fy_prevl, fy_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    }

    void CUDAForceAtlas2::freeGPUMemory()
    {
        hipFree(childl);

        hipFree(body_massl);
        hipFree(node_massl);
        hipFree(body_posl);
        hipFree(node_posl);
        hipFree(sourcesl);
        hipFree(targetsl);
        hipFree(countl);
        hipFree(startl);
        hipFree(sortl);

        hipFree(fxl);
        hipFree(fx_prevl);
        hipFree(fyl);
        hipFree(fy_prevl);

        hipFree(maxxl);
        hipFree(maxyl);
        hipFree(minxl);
        hipFree(minyl);

        hipFree(swgl);
        hipFree(etral);
    }

    CUDAForceAtlas2::~CUDAForceAtlas2()
    {
        free(body_mass);
        free(body_pos);
        free(sources);
        free(targets);
        free(fx);
        free(fy);
        free(fx_prev);
        free(fy_prev);

        freeGPUMemory();
    }

    void CUDAForceAtlas2::doStep()
    {
		/* Gravity kernel does not require BH tree. */
        GravityKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, k_g, strong_gravity, body_massl, body_posl, fxl, fyl);

        /* Attraction Kernel works based on edges. */
        if(this->use_linlog) {
            AttractiveLinLogForceKernel<<<mp_count * FACTOR6, THREADS6>>>(nedges, body_posl, fxl, fyl, sourcesl, targetsl);
        }
        else {
            AttractiveForceKernel<<<mp_count * FACTOR6, THREADS6>>>(nedges, body_posl, fxl, fyl, sourcesl, targetsl);
        }

		/* What does the BoundingBoxKernel do? */
        BoundingBoxKernel<<<mp_count * FACTOR1, THREADS1>>>(nnodes, nbodies, startl, childl, node_massl, body_posl, node_posl, maxxl, maxyl, minxl, minyl);

        // Build Barnes-Hut Tree
        // 1.) Set all child pointers of internal nodes (in childl) to null (-1)
        ClearKernel1<<<mp_count, 1024>>>(nnodes, nbodies, childl);
        // 2.) Build the tree
        TreeBuildingKernel<<<mp_count * FACTOR2, THREADS2>>>(nnodes, nbodies, childl, body_posl, node_posl);
        // 3.) Set all cell mass values to -1.0, set all startd to null (-1)
        ClearKernel2<<<mp_count, 1024>>>(nnodes, startl, node_massl);

        // Recursively compute mass for each BH. cell.
        SummarizationKernel<<<mp_count * FACTOR3, THREADS3>>>(nnodes, nbodies, countl, childl, body_massl, node_massl, body_posl, node_posl);

		/* No comments left for SortKernel. */
        SortKernel<<<mp_count * FACTOR4, THREADS4>>>(nnodes, nbodies, sortl, countl, startl, childl);

        // Compute repulsive forces between nodes using BH. tree.
        ForceCalculationKernel<<<mp_count * FACTOR5, THREADS5>>>(nnodes, nbodies, itolsq, epssq, sortl, childl, body_massl, node_massl, body_posl, node_posl, fxl, fyl, k_r);

        SpeedKernel<<<mp_count * FACTOR1, THREADS1>>>(nbodies, fxl, fyl, fx_prevl, fy_prevl, body_massl, swgl, etral);

        DisplacementKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, body_posl, fxl, fyl, fx_prevl, fy_prevl);

        cudaCatchError(hipDeviceSynchronize());
        iteration++;
    }

    void CUDAForceAtlas2::retrieveLayoutFromGPU()
    {
        cudaCatchError(hipMemcpy(body_pos, body_posl, sizeof(float2) * nbodies, hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
    }

    void CUDAForceAtlas2::sendLayoutToGPU()
    {
        cudaCatchError(hipMemcpy(body_posl, body_pos, sizeof(float2) * nbodies, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
    }

    void CUDAForceAtlas2::sendGraphToGPU()
    {
        cudaCatchError(hipMemcpy(body_massl, body_mass, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(sourcesl, sources, sizeof(int) * nedges, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(targetsl, targets, sizeof(int) * nedges, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
    }

	/**
	 * When is sync_layout() called?
	 * 
	 * This is host code, not implemented as a kernel.
	 */
    void CUDAForceAtlas2::sync_layout()
    {
        retrieveLayoutFromGPU();
        for(contiguous_nid_t n = 0; n < layout.graph.num_nodes(); ++n)
        {
            layout.setX(n, body_pos[n].x);
            layout.setY(n, body_pos[n].y);
        }
    }
}
